#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define f_(i,j) f_[(i) + (j)*(m)]
#define Z(i,j) Z[(i) + (j)*m]

__constant__ float s_H[121];	



//B is a partition of the images with dimensions thread number + patchSize
//A our extended array with padding 

__global__ void Zcalc(float const * const A,float const * const B, float *Z,float const * const H,int patchSize,float filtsigma, int m, int n)
{	
	int x = blockDim.x * blockIdx.x + threadIdx.x; 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x_local=threadIdx.x;
	int y_local = threadIdx.y;
	int pad =(patchSize-1)/2;
	int dimension=blockDim.x+patchSize-1;
	extern __shared__ float Memory[];
	float *s_A=&Memory[0];//local block position in s_A
	float *g_A=&Memory[dimension*dimension];//blocks global potition in g_A
	if(x<m-2*pad && y<n-2*pad){
		s_A[x_local +y_local*dimension]=B[x_local+y_local*dimension];
		if(x_local>blockDim.x-patchSize){
			s_A[(x_local+patchSize-1) + y_local*dimension]=B[x_local+patchSize-1 +y_local*dimension];
		}
		if(y_local>blockDim.y-patchSize){
			s_A[x_local + (y_local+patchSize-1)*dimension]=B[x_local + (y_local+patchSize-1)*dimension];
		}
		if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
			s_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=B[x_local+patchSize-1 + (y_local+patchSize-1)*dimension];
		}

		g_A[x_local +y_local*dimension]=A[x+y*m];
		if(x_local>blockDim.x-patchSize){
			g_A[(x_local+patchSize-1) + y_local*dimension]=A[x+patchSize-1 +y*m];
		}
		if(y_local>blockDim.y-patchSize){
			g_A[x_local + (y_local+patchSize-1)*dimension]=A[x + (y+patchSize-1)*m];
		}
		if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
			g_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=A[x+patchSize-1 + (y+patchSize-1)*m];
		}
		__syncthreads();
	}
	if(x<m-2*pad && y<n-2*pad){
		int counter=0;
		float temp=0,FNij=0,z_local=Z(x+pad,y+pad);
		
		for(int i=pad;i<dimension-pad;i++){
			for(int j=pad;j<dimension-pad;j++){
				for(int p=-pad;p<=pad;p++){
					for(int l=-pad;l<=pad;l++){
						temp=(g_A[(x_local+pad +l)+(y_local+pad + p)*dimension]-s_A[(i+l) + (j+p)*dimension])*s_H[counter];
						FNij=FNij+temp*temp;
						counter++;
					}
				}
				z_local=z_local+expf(-(FNij/filtsigma));
				FNij=0;
				counter=0;
			}
		}
		Z[x+pad + (y+pad)*m]=z_local;
	}
}

__global__ void fCalc(float const * const A,float const * const B,float const * const Z,float const * const H,float *f_,int patchSize, float filtsigma, int m, int n){
	int x = blockDim.x * blockIdx.x + threadIdx.x; 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x_local=threadIdx.x;
	int y_local = threadIdx.y;
	int pad =(patchSize-1)/2;
	int dimension=blockDim.x+patchSize-1;
	extern __shared__ float Memory[];
	float *s_A=&Memory[0];//local block position in s_A
	float *g_A=&Memory[dimension*dimension];//blocks global position g_A
	if(x<m-2*pad && y<n-2*pad){
		s_A[x_local +y_local*dimension]=B[x_local+y_local*dimension];
		if(x_local>blockDim.x-patchSize){
			s_A[(x_local+patchSize-1) + y_local*dimension]=B[x_local+patchSize-1 +y_local*dimension];
		}
		if(y_local>blockDim.y-patchSize){
			s_A[x_local + (y_local+patchSize-1)*dimension]=B[x_local + (y_local+patchSize-1)*dimension];
		}
		if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
			s_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=B[x_local+patchSize-1 + (y_local+patchSize-1)*dimension];
		}
		g_A[x_local +y_local*dimension]=A[x+y*m];
		if(x_local>blockDim.x-patchSize){
			g_A[(x_local+patchSize-1) + y_local*dimension]=A[x+patchSize-1 +y*m];
		}
		if(y_local>blockDim.y-patchSize){
			g_A[x_local + (y_local+patchSize-1)*dimension]=A[x + (y+patchSize-1)*m];
		}
		if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
			g_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=A[x+patchSize-1 + (y+patchSize-1)*m];
		}
		__syncthreads();
	}
	if(x<m-2*pad && y<n-2*pad){
		int i,j,k,l,counter=0;
		float temp=0,FNij=0,Z_local=Z(x+pad,y+pad),f_local=f_(x+pad,y+pad);
		
		for(i=pad;i<dimension-pad;i++){
			for(j=pad;j<dimension-pad;j++){
				for(k=-pad;k<=pad;k++){
					for(l=-pad;l<=pad;l++){
						temp=(g_A[(x_local+pad +l)+(y_local+pad + k)*dimension]-s_A[(i+l) + (j+k)*dimension])*s_H[counter];
						FNij=FNij+temp*temp;
						counter++;
					}
				}
				f_local=f_local+(1/Z_local)*(expf(-(FNij/filtsigma)))*s_A[i+j*dimension];
				FNij=0;
				counter=0;
			}
		}
		f_[x+pad + (y+pad)*m]=f_local;
	}
}
/*
__global__ void Zcalc(float const *const A,float const * const B,float const * const H,float *Z,int patchSize,float filtSigma,int m , int n)
{ //global coordinates
  int x=blockIdx.x*blockDim.x + threadIdx.x;
  int y=blockIdx.y*blockDim.y + threadIdx.y;
  //block coordinates
  int xblock=threadIdx.x;
  int yblock=threadIdx.y;
  extern __shared__ float Memory[];
  int dimension=blockDim.x + patchSize-1;
  
  float *s_A=&Memory[0];
  float *g_A=&Memory[dimension*dimension];
  if(x<m-patchSize+1 && y<n-patchSize+1)
  {
    s_A[xblock + yblock*dimension]=B[xblock+yblock*dimension];
    __syncthreads();
    if(xblock>blockDim.x-patchSize){
      s_A[xblock+patchSize-1+ yblock*dimension]=B[xblock+patchSize-1 + yblock*dimension];
    }
    __syncthreads();    
    if(yblock>blockDim.y-patchSize){
	  s_A[xblock+ (yblock+patchSize-1)*dimension ]=B[xblock+ (yblock+patchSize-1)*dimension ];
	}
	__syncthreads();
    if(xblock>blockDim.x-patchSize && yblock>blockDim.y-patchSize){
	    s_A[xblock+patchSize-1 + (yblock+patchSize-1)*dimension]=B[xblock+patchSize-1 + (yblock+patchSize-1)*dimension];
	}
	__syncthreads();
    //global coordinates of block
    g_A[xblock + yblock*dimension]=A[x + y*m];
    __syncthreads();
    if(xblock>blockDim.x-patchSize){
      g_A[xblock+patchSize-1 + yblock*dimension]=A[x+patchSize-1 + y*m];
    }
    __syncthreads();
    if(yblock>blockDim.y-patchSize){
	  g_A[xblock+ (yblock+patchSize-1)*dimension ]=A[x+ (y+patchSize-1)*m ];
	}
	__syncthreads();
    if(xblock>blockDim.x-patchSize && yblock>blockDim.y-patchSize){
	    g_A[xblock+patchSize-1 + (yblock+patchSize-1)*dimension]=A[x+patchSize-1 + (y+patchSize-1)*m];
	}
  }
    __syncthreads();
    patchSize=(patchSize-1)/2;

    if(x<m-2*patchSize && y<n-2*patchSize)
    {
	  int i,j,k,l,counter=0;
	  float FNij=0,temp=0;
	  float Z_local=Z(x+patchSize,y+patchSize);
	  for(i=patchSize;i<dimension-patchSize;i++){
	    for(j=patchSize;j<dimension-patchSize;j++){
		  for(k=-patchSize;k<=patchSize;k++){
		    for(l=-patchSize;l<=patchSize;l++){
		      temp=(g_A[xblock+patchSize+l + (yblock+patchSize+k)*dimension]-s_A[i+l + (j+k)*dimension])*H[counter];
		      temp=temp*temp;
		      FNij=FNij+temp;
		      counter++;
		    }
		  }
		  Z_local=Z_local+expf(- (FNij/filtSigma));
		  FNij=0;
		  counter=0;
		}
	  }
	 Z(x+patchSize,y+patchSize)=Z_local;
	}
}

__global__ void fCalc(float const *const A,float const * const B,float const * const H,float const * const Z,float *f_,int patchSize,float filtSigma,int m , int n)
{ //global coordinates
  int x=blockIdx.x*blockDim.x + threadIdx.x;
  int y=blockIdx.y*blockDim.y + threadIdx.y;
  //block coordinates
  int xblock=threadIdx.x;
  int yblock=threadIdx.y;
  extern __shared__ float Memory[];
  int dimension=blockDim.x + patchSize-1;
  
  float *s_A=&Memory[0];
  float *g_A=&Memory[dimension*dimension];
   if(x<m-patchSize+1 && y<n-patchSize+1)
  {
    s_A[xblock + yblock*dimension]=B[xblock+yblock*dimension];
    __syncthreads();
    if(xblock>blockDim.x-patchSize){
      s_A[xblock+patchSize-1+ yblock*dimension]=B[xblock+patchSize-1 + yblock*dimension];
    }
    __syncthreads();    
    if(yblock>blockDim.y-patchSize){
	  s_A[xblock+ (yblock+patchSize-1)*dimension ]=B[xblock+ (yblock+patchSize-1)*dimension ];
	}
	__syncthreads();
    if(xblock>blockDim.x-patchSize && yblock>blockDim.y-patchSize){
	    s_A[xblock+patchSize-1 + (yblock+patchSize-1)*dimension]=B[xblock+patchSize-1 + (yblock+patchSize-1)*dimension];
	}
	__syncthreads();
    //global coordinates of block
    g_A[xblock + yblock*dimension]=A[x + y*m];
    __syncthreads();
    if(xblock>blockDim.x-patchSize){
      g_A[xblock+patchSize-1 + yblock*dimension]=A[x+patchSize-1 + y*m];
    }
    __syncthreads();
    if(yblock>blockDim.y-patchSize){
	  g_A[xblock+ (yblock+patchSize-1)*dimension ]=A[x+ (y+patchSize-1)*m ];
	}
	__syncthreads();
    if(xblock>blockDim.x-patchSize && yblock>blockDim.y-patchSize){
	    g_A[xblock+patchSize-1 + (yblock+patchSize-1)*dimension]=A[x+patchSize-1 + (y+patchSize-1)*m];
	}
  }
    __syncthreads();
  patchSize=(patchSize-1)/2;
  if(x<m-2*patchSize && y<n-2*patchSize)
  {
    int i,j,k,l,counter=0;
	  float FNij=0,temp=0;
	  float Z_local=Z(x+patchSize,y+patchSize),f_local=f_(x+patchSize,x+patchSize);
	  for(i=patchSize;i<dimension-patchSize;i++){
	    for(j=patchSize;j<dimension-patchSize;j++){
		  for(k=-patchSize;k<=patchSize;k++){
		    for(l=-patchSize;l<=patchSize;l++){
		      temp=(g_A[xblock+patchSize+l + (yblock+patchSize+k)*dimension]-s_A[i+l + (j+k)*dimension])*H[counter];
		      temp=temp*temp;
		      FNij=FNij+temp;
		      counter++;
		    }
		  }
		  f_local=f_local+(1/Z_local)*(expf(- (FNij/filtSigma)))*(s_A[i+(j)*dimension]);
		  FNij=0;
		  counter=0;
		}
	  }
    f_(x+patchSize,y+patchSize)=f_local;
  }
}
*/
