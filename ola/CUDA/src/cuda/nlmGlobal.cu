#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
// Array access macros
#define f(i,j) A[(i) + (j)*(m)]
#define B(i,j) B[(i) + (j)*(m)]
#define Z(x,y) Z[(x) + (y)*(m)]
#define f_(x,y) f_[(x) + (y)*(m)]


__global__ void Zcalc(float const * const A, float *Z,float const * const H,int patchSize,float patchSigma,float fltSigma, int m, int n) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x<m-(2*patchSize-1)/2 && y<n-(2*patchSize-1)/2){
    
    int i,j,k,l,counter=0;
    float FNij=0.0;
    float temp=0.0;
    patchSize=(patchSize-1)/2;
    for(i=patchSize;i<m-patchSize;i++){
	  for(j=patchSize;j<n-patchSize;j++){
		for(k=-patchSize;k<=patchSize;k++){
		  for(l=-patchSize;l<=patchSize;l++){
			temp=(f(x+patchSize+k,y+patchSize+l)-f(i+k,j+l))*H[counter];
			temp=temp*temp;
            FNij=FNij+(temp);
            counter++;
		  }
		}
	    Z(x+patchSize,y+patchSize)=Z(x+patchSize,y+patchSize)+expf(-(FNij/(fltSigma)));
	    FNij=0.0;
	    counter=0;
	  }
    }
  }
}

__global__ void fCalc(float const * const A,float const * const Z,float const * const H, float *f_,int patchSize,float patchSigma,float fltSigma, int m, int n){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x<m-(2*patchSize-1)/2 && y<n-(2*patchSize-1)/2){
	int i,j,k,l,counter=0;
    patchSize=(patchSize-1)/2;
    float FNij=0.0;
    float temp=0.0;
    float Z_local=Z(x+patchSize,y+patchSize);
    for(i=patchSize;i<m-patchSize;i++){
	  for(j=patchSize;j<n-patchSize;j++){
		for(k=-patchSize;k<=patchSize;k++){
		  for(l=-patchSize;l<=patchSize;l++){
			temp=(f(x+patchSize+k,y+patchSize+l)-f(i+k,j+l))*H[counter];
			temp=temp*temp;
            FNij=FNij+(temp);
            counter++;
		  }
		}
	    f_(x+patchSize,y+patchSize)=f_(x+patchSize,y+patchSize)+((1/Z_local)*expf(-(FNij/(fltSigma))))*f(i,j);
	    FNij=0.0;
	    counter=0;
	  }
    }
  }
}


