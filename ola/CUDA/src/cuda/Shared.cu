#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


#define f(x,y) A[x + (y)* m]
#define Z(x,y) Z[x + (y)* m]
#define f_(x,y) f_[(x) + (y)*m]



__constant__ float s_H[81];	



//B is a partition of the images with dimensions thread number + patchSize
//A our extended array with padding 

__global__ void Zcalc(float const * const A, float *Z,int patchSize,float filtsigma, int m, int n)
{	
	int x = blockDim.x * blockIdx.x + threadIdx.x; 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x_local=threadIdx.x;
	int y_local = threadIdx.y;
	int pad =(patchSize-1)/2;
	int dimension=blockDim.x+patchSize-1;
	extern __shared__ float Memory[];
	float z_local=0;
	float *s_A=&Memory[0];//local block position in s_A
	float *g_A=&Memory[dimension*dimension];//block's global position in g_A
	if(x<m-2*pad && y<n-2*pad){
	  z_local=Z(x+pad,y+pad);
	  g_A[x_local +y_local*dimension]=A[x+y*m];
	  if(x_local>blockDim.x-patchSize){		  
		g_A[(x_local+patchSize-1) + y_local*dimension]=A[x+patchSize-1 +y*m];
	  }
	  if(y_local>blockDim.y-patchSize){
	  g_A[x_local + (y_local+patchSize-1)*dimension]=A[x + (y+patchSize-1)*m];
	  }
	  if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
		g_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=A[x+patchSize-1 + (y+patchSize-1)*m];
      }
	}
    for(int xpos=0;xpos<(m-patchSize+1);xpos=xpos+blockDim.x){
	for(int ypos=0;ypos<(n-patchSize+1);ypos=ypos+blockDim.y){	
    __syncthreads();
	if(x<m-2*pad && y<n-2*pad){
		s_A[x_local +y_local*dimension]=A[x_local+xpos+(y_local+ypos)*m];
		if(x_local>blockDim.x-patchSize){
			s_A[(x_local+patchSize-1) + y_local*dimension]=A[x_local+xpos+patchSize-1 +(y_local+ypos)*m];
		}
		if(y_local>blockDim.y-patchSize){
			s_A[x_local + (y_local+patchSize-1)*dimension]=A[x_local +xpos + (y_local+ypos+patchSize-1)*m];
		}
		if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
			s_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=A[x_local+xpos+patchSize-1 + (y_local+ypos+patchSize-1)*m];
		}
	}
	__syncthreads();
	if(x<m-2*pad && y<n-2*pad){
		int counter=0;
		float temp=0,FNij=0;
		
		for(int i=pad;i<dimension-pad;i++){
			for(int j=pad;j<dimension-pad;j++){
				for(int p=-pad;p<=pad;p++){
					for(int l=-pad;l<=pad;l++){
						temp=(g_A[(x_local+pad +l)+(y_local+pad + p)*dimension]-s_A[(i+l) + (j+p)*dimension])*s_H[counter];
						FNij=FNij+temp*temp;
						counter++;
					}
				}
				z_local=z_local+expf(-(FNij/filtsigma));
				FNij=0;
				counter=0;
			}
		}
	}
}
}
  Z[x+pad + (y+pad)*m]=z_local;
}

__global__ void fCalc(float const * const A,float const * const Z,float *f_,int patchSize, float filtsigma, int m, int n){
	int x = blockDim.x * blockIdx.x + threadIdx.x; 
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int x_local=threadIdx.x;
	int y_local = threadIdx.y;
	int pad =(patchSize-1)/2;
	int dimension=blockDim.x+patchSize-1;
	extern __shared__ float Memory[];
	float Z_local=0, f_local=0;
	float *s_A=&Memory[0];//local block position in s_A
	float *g_A=&Memory[dimension*dimension];//blocks global position g_A
    if(x<m-2*pad && y<n-2*pad){
	  Z_local=Z(x+pad,y+pad),f_local=f_(x+pad,y+pad);
      g_A[x_local +y_local*dimension]=A[x+y*m];
      if(x_local>blockDim.x-patchSize){
	    g_A[(x_local+patchSize-1) + y_local*dimension]=A[x+patchSize-1 +y*m];
	  }
	  if(y_local>blockDim.y-patchSize){
	    g_A[x_local + (y_local+patchSize-1)*dimension]=A[x + (y+patchSize-1)*m];
	  }
	  if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
	    g_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=A[x+patchSize-1 + (y+patchSize-1)*m];
	  }
    }
	for(int xpos=0;xpos<(m-patchSize+1);xpos=xpos+blockDim.x){
	for(int ypos=0;ypos<(n-patchSize+1);ypos=ypos+blockDim.y){
	__syncthreads();
	if(x<m-2*pad && y<n-2*pad){
		s_A[x_local+ +y_local*dimension]=A[x_local+xpos+(y_local+ypos)*m];
		if(x_local>blockDim.x-patchSize){
			s_A[(x_local+patchSize-1) + y_local*dimension]=A[x_local+xpos+patchSize-1 +(y_local+ypos)*m];
		}
		if(y_local>blockDim.y-patchSize){
			s_A[x_local + (y_local+patchSize-1)*dimension]=A[x_local+xpos + (y_local+ypos+patchSize-1)*m];
		}
		if(x_local>blockDim.x-patchSize && y_local>blockDim.y-patchSize ){
			s_A[x_local+patchSize-1 + (y_local+patchSize-1)*dimension]=A[x_local+xpos+patchSize-1 + (y_local+ypos+patchSize-1)*m];
		}
		__syncthreads();
	}
	if(x<m-2*pad && y<n-2*pad){
		int i,j,k,l,counter=0;
		float temp=0,FNij=0;
		for(i=pad;i<dimension-pad;i++){
			for(j=pad;j<dimension-pad;j++){
				for(k=-pad;k<=pad;k++){
					for(l=-pad;l<=pad;l++){
						temp=(g_A[(x_local+pad +l)+(y_local+pad + k)*dimension]-s_A[(i+l) + (j+k)*dimension])*s_H[counter];
						FNij=FNij+temp*temp;
						counter++;
					}
				}
				f_local=f_local+(1/Z_local)*(expf(-(FNij/filtsigma)))*s_A[i+j*dimension];
				FNij=0;
				counter=0;
			}
		}
	}
}
}
  f_[x+pad + (y+pad)*m]=f_local;
}
